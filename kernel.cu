
#include <hip/hip_runtime.h>
// Vector addition (device code)

// extern C for host program load correct function name
extern "C" __global__ void Sum(int *a, int *b, int *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n)
        c[tid] = a[tid] + b[tid];
}
