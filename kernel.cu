
#include <hip/hip_runtime.h>
// Vector addition (device code)

extern "C" __global__ void Sum(int *a, int *b, int *c, int n)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n)
        c[tid] = a[tid] + b[tid];
}
